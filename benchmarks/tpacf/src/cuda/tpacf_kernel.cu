#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

#include <assert.h>
#include "model.h"
#include <math.h>

#define WARP_SIZE 32
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#define BLOCK_SIZE 256
#define NUM_WARPS (BLOCK_SIZE/WARP_SIZE)
#define HISTS_PER_WARP 16
#define NUM_HISTOGRAMS  (NUM_WARPS*HISTS_PER_WARP)
#define THREADS_PER_HIST (WARP_SIZE/HISTS_PER_WARP)

REAL** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;

__constant__ REAL dev_binb[NUM_BINS+1];

unsigned int NUM_SETS;
unsigned int NUM_ELEMENTS;

// create the bin boundaries
void initBinB( struct pb_TimerSet *timers )
{
  REAL *binb = (REAL*)malloc((NUM_BINS+1)*sizeof(REAL));
  for (int k = 0; k < NUM_BINS+1; k++)
    {
      binb[k] = cos(pow(10.0, (log10(min_arcmin) + k*1.0/bins_per_dec)) 
		    / 60.0*D2R);
    }
  pb_SwitchToTimer( timers, pb_TimerID_COPY );
  hipMemcpyToSymbol(HIP_SYMBOL(dev_binb), binb, (NUM_BINS+1)*sizeof(REAL));
  pb_SwitchToTimer( timers, pb_TimerID_COMPUTE );
  free(binb);
}

__global__ 
void gen_hists( hist_t* histograms, REAL* all_x_data, REAL* all_y_data, 
		REAL* all_z_data, int NUM_SETS, int NUM_ELEMENTS )
{
  unsigned int bx = blockIdx.x;
  unsigned int tid = threadIdx.x;
  bool do_self = (bx < (NUM_SETS + 1));

  REAL* data_x;
  REAL* data_y;
  REAL* data_z;
  REAL* random_x;
  REAL* random_y;
  REAL* random_z;

  __shared__ struct cartesian data_s[BLOCK_SIZE];
  
  __shared__ unsigned int 
    warp_hists[NUM_BINS][NUM_HISTOGRAMS]; // 640B <1k  
   
	//initialize all warp_hists block
	//initialization is intrinsic by default
	__asm__("INTRN:");	 
  for(unsigned int w = 0; w < NUM_BINS*NUM_HISTOGRAMS; w += BLOCK_SIZE )
    {
	__asm__("INTRN:");	 
      if(w+tid < NUM_BINS*NUM_HISTOGRAMS)
	{
	  warp_hists[(w+tid)/NUM_HISTOGRAMS][(w+tid)%NUM_HISTOGRAMS] = 0;
	}
	__asm__("INTRN:");	 
    }
    

 // Get stuff into shared memory to kick off the loop.

	//Again: data structure initialization based on 
	//thread id. This is intrinsic
	__asm__("INTRN:");	 
  if( !do_self)
    {
      data_x = all_x_data;
      data_y = all_y_data;
      data_z = all_z_data;
      random_x = all_x_data + NUM_ELEMENTS * (bx - NUM_SETS);
      random_y = all_y_data + NUM_ELEMENTS * (bx - NUM_SETS);
      random_z = all_z_data + NUM_ELEMENTS * (bx - NUM_SETS);
    }
	__asm__("INTRN:");	 
  else
    {
      random_x = all_x_data + NUM_ELEMENTS * (bx);
      random_y = all_y_data + NUM_ELEMENTS * (bx);
      random_z = all_z_data + NUM_ELEMENTS * (bx);
      
      data_x = random_x;
      data_y = random_y;
      data_z = random_z;
    }
    
  // Iterate over all data points
	__asm__("EXTRN:");	//data bound check, extrinsic
	//This for loops is a data loading loop
  for(unsigned int i = 0; i < NUM_ELEMENTS; i += BLOCK_SIZE )
    {
      // load current set of data into shared memory
      // (total of BLOCK_SIZE points loaded)
	__asm__("EXTRN:");	//data bound check, extrinsic	
      if( tid + i < NUM_ELEMENTS )
	{ // reading outside of bounds is a-okay
	  data_s[tid] = (struct cartesian)
            {data_x[tid + i], data_y[tid + i], data_z[tid + i]};
	}
      
      __syncthreads();

      // Iterate over all random points

	//TODO: Conditional statement in for initialization
	__asm__("EXTRN:");	//data bound check, extrinsic	
      for(unsigned int j = (do_self ? i+1 : 0); j < NUM_ELEMENTS; 
	  j += BLOCK_SIZE)
	{
	  // load current random point values
	  REAL random_x_s;
	  REAL random_y_s;
	  REAL random_z_s;
	  
	__asm__("EXTRN:");	//data bound check, extrinsic	
	  if(tid + j < NUM_ELEMENTS)
	    {
	      random_x_s = random_x[tid + j];
	      random_y_s = random_y[tid + j];
	      random_z_s = random_z[tid + j];
	    }

	  // Iterate for all elements of current set of data points 
	  // (BLOCK_SIZE iterations per thread)
	  // Each thread calcs against 1 random point within cur set of random
	  // (so BLOCK_SIZE threads covers all random points within cur set)
	//TODO: check tow cconsitions in for
	__asm__("EXTRN:");	//data bound check, extrinsic	
	__asm__("EXTRN:");	//data bound check, extrinsic	
	  for(unsigned int k = 0; 
	      (k < BLOCK_SIZE) && (k+i < NUM_ELEMENTS);
	      k += 1)
	    {
	      // do actual calculations on the values:
	      REAL distance = 
	        data_s[k].x * random_x_s +
		    data_s[k].y * random_y_s +
		    data_s[k].z * random_z_s;

	      unsigned int bin_index;

	      // run binary search to find bin_index
	      unsigned int min = 0;
	      unsigned int max = NUM_BINS;
		//binary search, all branches intrinsic
	      {
		unsigned int k2;
	      	
		__asm__("INTRN:");	//TODO: (Check) for the while
		while (max > min+1)
		  {
		    k2 = (min + max) / 2;
			__asm__("INTRN:");//if
		    if (distance >= dev_binb[k2]) 
		      max = k2;
			__asm__("INTRN:");//else
		    else 
		      min = k2;
		__asm__("INTRN:");	//for the while
		  }
		bin_index = max - 1;
	      }

	      unsigned int warpnum = tid / (WARP_SIZE/HISTS_PER_WARP);
		//TODO; complicated if statement
		__asm__("INTRN:");
		__asm__("INTRN:");
		__asm__("INTRN:");
		__asm__("INTRN:");
		__asm__("EXTRN:");
	      if((distance < dev_binb[min]) &&	//intrinsic 
		(distance >= dev_binb[max]) && 	//intrinsic
		//TODO: check if or creates 2 conditions
		(!do_self || 			//intrinsic
		(tid + j > i + k)) && 		//intrinsic
		(tid + j < NUM_ELEMENTS))	//extrinsic
		{
		  atomicAdd(&warp_hists[bin_index][warpnum], 1U);
		}
		//TODO: check two consitions for for
		__asm__("EXTRN:");	//data bound check, extrinsic	
		__asm__("EXTRN:");	//data bound check, extrinsic	
	    }
	__asm__("EXTRN:");	//end of for/data bound check, extrinsic	
	}
	__asm__("EXTRN:");	//end of first for, 	
				//data bound check, extrinsic
    }
    
  // coalesce the histograms in a block
  unsigned int warp_index = tid & ( (NUM_HISTOGRAMS>>1) - 1);
  unsigned int bin_index = tid / (NUM_HISTOGRAMS>>1);
	__asm__("INTRN:");	//intrinsic because
	//the loop must continue until the offset is nil
  for(unsigned int offset = NUM_HISTOGRAMS >> 1; offset > 0; 
      offset >>= 1)
    {
	__asm__("EXTRN:")	//data bound check
      for(unsigned int bin_base = 0; bin_base < NUM_BINS; 
	  bin_base += BLOCK_SIZE/ (NUM_HISTOGRAMS>>1))
	{
	  __syncthreads();

		__asm__("INTRN:");	//check if the current
		//warp is meant to fill this bin of historgram
		__asm__("EXTRN:");	//data bound check	
	  if(warp_index < offset 	//intrinsic
		&& bin_base+bin_index < NUM_BINS )	//extrinsic
	    {
	      unsigned long sum =
		warp_hists[bin_base + bin_index][warp_index] + 
		warp_hists[bin_base + bin_index][warp_index+offset];
	      warp_hists[bin_base + bin_index][warp_index] = sum;
	    }
	__asm__("EXTRN:")	//end of for, data bound check
	}
	__asm__("INTRN:");	//end of previous for, intrinsic
    }
    
  __syncthreads();
    
  // Put the results back in the real histogram
  // warp_hists[x][0] holds sum of all locations of bin x
  hist_t* hist_base = histograms + NUM_BINS * bx;
	__asm__("EXTRN:");	//bound check on number of BINS
				//i.e. no. of pillars in histogram
				//extrinsic
  if(tid < NUM_BINS)
    {
      hist_base[tid] = warp_hists[tid][0];
    }
}

void TPACF(hist_t * histograms, REAL* d_x_data, REAL* d_y_data, 
	   REAL* d_z_data)
{
  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid(NUM_SETS*2 + 1);

  gen_hists <<< dimGrid, dimBlock >>> ( histograms, d_x_data, 
					d_y_data, d_z_data, NUM_SETS, 
					NUM_ELEMENTS);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
